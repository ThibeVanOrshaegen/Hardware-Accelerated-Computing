#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../Include/stb_image_write.h"
#define STB_IMAGE_IMPLEMENTATION
#include "../Include/stb_image.h"
#include <time.h>

clock_t start, stop;
double cpu_time;

#define BLOCK_SIZE 128

__global__ void applyConvolution(unsigned char* image, unsigned char* output, int width, int height, int channels, float* kernel) {
    int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    int x = threadid % width;
    int y = threadid / width;
    int edge = 1; // Since kernel size is 3x3

    while (y < height) {
        float sum[3] = {0.0, 0.0, 0.0}; // Sum for each channel
        /*Dit zou in een kernel moeten*/
        for (int ky = -edge; ky <= edge; ky++) {
            for (int kx = -edge; kx <= edge; kx++) {
                int ix = x + kx;
                int iy = y + ky;
                if (ix >= 0 && ix < width && iy >= 0 && iy < height) {
                    for (int ch = 0; ch < channels; ch++) {
                        if (ch < 3) { // Apply convolution only to RGB channels
                            sum[ch] += kernel[(ky + edge) * 3 + (kx + edge)] * image[(iy * width + ix) * channels + ch];
                        }
                    }
                }
            }
        }
        for (int ch = 0; ch < channels; ch++) {
            if (ch < 3) {
                int val = (int)sum[ch];
                output[(y * width + x) * channels + ch] = (unsigned char)(val > 255 ? 255 : (val < 0 ? 0 : val));
            } else {
                // Preserve the alpha channel if present
                output[(y * width + x) * channels + ch] = 255;
            }
        }

        threadid += blockDim.x * gridDim.x;
        x = threadid % width;
        y = threadid / width;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        printf("Usage: %s <image_path1> <image_path2> ... <image_pathN>\n", argv[0]);
        return 1;
    }

    int width, height, channels;
    int imageCount = argc - 1;

    for (int i = 0; i < imageCount; ++i) {
        unsigned char* img = stbi_load(argv[i + 1], &width, &height, &channels, 0);
        if (img == NULL) {
            printf("Error in loading the image: %s\n", argv[i + 1]);
            continue;
        }

        unsigned char* grayImg = (unsigned char*)malloc(width * height * channels);

        for (int j = 0; j < width * height; j++) {
            grayImg[j] = (unsigned char)(0.2989 * img[3 * j] + 0.5870 * img[3 * j + 1] + 0.1140 * img[3 * j + 2]);
        }

        float kernel[3] = {
            1, 0, -1,
            1, 0, -1,
            1, 0, -1
        };

        unsigned char* outputImg = (unsigned char*)malloc(width * height * channels);

        start = clock();

        unsigned char* d_img;
        hipMalloc(&d_img, width * height * channels);
        hipMemcpy(d_img, grayImg, width * height * channels, hipMemcpyHostToDevice);

        unsigned char* d_outputImg;
        hipMalloc(&d_outputImg, width * height * channels);

        float* d_kernel;
        hipMalloc(&d_kernel, 3 * 3 * sizeof(float));
        hipMemcpy(d_kernel, kernel, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);

        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

        applyConvolution<<<1023, 256>>>(d_img, d_outputImg, width, height, channels, d_kernel);

        hipDeviceSynchronize();
        hipMemcpy(outputImg, d_outputImg, width * height * channels, hipMemcpyDeviceToHost);

        stop = clock();
        cpu_time = ((double)(stop - start)) / CLOCKS_PER_SEC;
        printf("Time taken for image %d: %f\n", i + 1, cpu_time);

        char OutputPath[100];
        snprintf(OutputPath, sizeof(OutputPath), "%s-output.png", argv[i + 1]);
        stbi_write_png(OutputPath, width, height, channels, outputImg, width * channels);

        stbi_image_free(img);
        free(outputImg);
        free(grayImg);

        hipFree(d_img);
        hipFree(d_outputImg);
        hipFree(d_kernel);
    }

    return 0;
}
